#include "hip/hip_runtime.h"

// This file is part of the LITIV framework; visit the original repository at
// https://github.com/plstcharles/litiv for more information.
//
// Copyright 2017 Pierre-Luc St-Charles; pierre-luc.st-charles<at>polymtl.ca
//
// Licensed under the Apache License, Version 2.0 (the "License");
// you may not use this file except in compliance with the License.
// You may obtain a copy of the License at
//     http://www.apache.org/licenses/LICENSE-2.0
//
// Unless required by applicable law or agreed to in writing, software
// distributed under the License is distributed on an "AS IS" BASIS,
// WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
// See the License for the specific language governing permissions and
// limitations under the License.
//

#include "test.cuh"

__global__ void device::test(int n) {
    const int px = blockIdx.x*blockDim.x + threadIdx.x;
    const int py = blockIdx.y*blockDim.y + threadIdx.y;
    const int pz = blockIdx.z*blockDim.z + threadIdx.z;
    printf("cuda test kernel @ px = %d, py = %d, pz = %d, with n = %d\n",px,py,pz,n);
}

void host::test(const lv::cuda::KernelParams& oKParams, int n) {
    cudaKernelWrap(test,oKParams,n);
}

// for use via extern in litiv/utils/cuda.hpp
namespace lv {
    namespace cuda {
        void test(const lv::cuda::KernelParams& oKParams, int n) {
            host::test(oKParams,n);
        }
    }
}