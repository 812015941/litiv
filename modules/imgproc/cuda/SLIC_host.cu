#include "hip/hip_runtime.h"

// This file is part of the LITIV framework; visit the original repository at
// https://github.com/plstcharles/litiv for more information.
//
// Copyright 2017 Pierre-Luc St-Charles; pierre-luc.st-charles<at>polymtl.ca
//
// Licensed under the Apache License, Version 2.0 (the "License");
// you may not use this file except in compliance with the License.
// You may obtain a copy of the License at
//     http://www.apache.org/licenses/LICENSE-2.0
//
// Unless required by applicable law or agreed to in writing, software
// distributed under the License is distributed on an "AS IS" BASIS,
// WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
// See the License for the specific language governing permissions and
// limitations under the License.
//
// //////////////////////////////////////////////////////////////////////////
//
//               SLIC Superpixel Oversegmentation Algorithm
//       CUDA implementation of Achanta et al.'s method (TPAMI 2012)
//
// Note: requires CUDA compute architecture >= 3.0
// Author: Francois-Xavier Derue
// Contact: francois.xavier.derue@gmail.com
// Source: https://github.com/fderue/SLIC_CUDA
//
// Copyright (c) 2016 fderue
//
// Permission is hereby granted, free of charge, to any person obtaining a copy
// of this software and associated documentation files (the "Software"), to deal
// in the Software without restriction, including without limitation the rights
// to use, copy, modify, merge, publish, distribute, sublicense, and/or sell
// copies of the Software, and to permit persons to whom the Software is
// furnished to do so, subject to the following conditions:
//
// The above copyright notice and this permission notice shall be included in all
// copies or substantial portions of the Software.
//
// THE SOFTWARE IS PROVIDED "AS IS", WITHOUT WARRANTY OF ANY KIND, EXPRESS OR
// IMPLIED, INCLUDING BUT NOT LIMITED TO THE WARRANTIES OF MERCHANTABILITY,
// FITNESS FOR A PARTICULAR PURPOSE AND NONINFRINGEMENT. IN NO EVENT SHALL THE
// AUTHORS OR COPYRIGHT HOLDERS BE LIABLE FOR ANY CLAIM, DAMAGES OR OTHER
// LIABILITY, WHETHER IN AN ACTION OF CONTRACT, TORT OR OTHERWISE, ARISING FROM,
// OUT OF OR IN CONNECTION WITH THE SOFTWARE OR THE USE OR OTHER DEALINGS IN THE
// SOFTWARE.
//

#include "litiv/imgproc/SLIC.hpp"
#include "SLIC_device.hpp"

using namespace std;
using namespace cv;

SLIC::SLIC(){
    int nbGpu = 0;
    cudaErrorCheck(hipGetDeviceCount(&nbGpu));
    cout << "Detected " << nbGpu << " cuda capable gpu" << endl;
    cudaErrorCheck(hipSetDevice(m_deviceId));
    cudaErrorCheck(hipGetDeviceProperties(&m_deviceProp, m_deviceId));
    if (m_deviceProp.major < 3){
        cerr << "compute capability found = " << m_deviceProp.major << ", compute capability >= 3 required !" << endl;
        exit(EXIT_FAILURE);
    }
}

SLIC::~SLIC(){
    delete[] h_fClusters;
    delete[] h_fLabels;
    cudaErrorCheck(hipFree(d_fClusters));
    cudaErrorCheck(hipFree(d_fAccAtt));
    cudaErrorCheck(hipFreeArray(cuArrayFrameBGRA));
    cudaErrorCheck(hipFreeArray(cuArrayFrameLab));
    cudaErrorCheck(hipFreeArray(cuArrayLabels));
}

void SLIC::initialize(const cv::Mat& frame0, const int diamSpxOrNbSpx , const InitType initType, const float wc , const int nbIteration ) {
    m_nbIteration = nbIteration;
    m_FrameWidth = frame0.cols;
    m_FrameHeight = frame0.rows;
    m_nbPx = m_FrameWidth*m_FrameHeight;
    m_InitType = initType;
    m_wc = wc;
    if (m_InitType == SLIC_NSPX){
        m_SpxDiam = diamSpxOrNbSpx;
        m_SpxDiam = (int)sqrt(m_nbPx / (float)diamSpxOrNbSpx);
    }
    else m_SpxDiam = diamSpxOrNbSpx;

    getSpxSizeFromDiam(m_FrameWidth, m_FrameHeight, m_SpxDiam, &m_SpxWidth, &m_SpxHeight); // determine w and h of Spx based on diamSpx
    m_SpxArea = m_SpxWidth*m_SpxHeight;
    CV_Assert(m_nbPx%m_SpxArea == 0);
    m_nbSpx = m_nbPx / m_SpxArea;

    h_fClusters = new float[m_nbSpx * 5]; // m_nbSpx * [L,a,b,x,y]
    h_fLabels = new float[m_nbPx];

    initGpuBuffers();
}

void SLIC::segment(const Mat& frameBGR) {
    uploadFrame(frameBGR);
    gpuRGBA2Lab();
    gpuInitClusters();

    for (int i = 0; i<m_nbIteration; i++) {
        assignment();
        hipDeviceSynchronize();
        update();
        hipDeviceSynchronize();
    }
    downloadLabels();
}

void SLIC::initGpuBuffers() {
    //allocate buffers on gpu

    hipChannelFormatDesc channelDescrBGRA = hipCreateChannelDesc(8, 8, 8, 8, hipChannelFormatKindUnsigned);
    cudaErrorCheck(hipMallocArray(&cuArrayFrameBGRA, &channelDescrBGRA, m_FrameWidth, m_FrameHeight));

    hipChannelFormatDesc channelDescrLab = hipCreateChannelDesc(32, 32, 32, 32, hipChannelFormatKindFloat);
    cudaErrorCheck(hipMallocArray(&cuArrayFrameLab, &channelDescrLab, m_FrameWidth, m_FrameHeight, hipArraySurfaceLoadStore));

    hipChannelFormatDesc channelDescrLabels = hipCreateChannelDesc(32, 0, 0, 0, hipChannelFormatKindFloat);
    cudaErrorCheck(hipMallocArray(&cuArrayLabels, &channelDescrLabels, m_FrameWidth, m_FrameHeight, hipArraySurfaceLoadStore));

    // Specify texture frameBGRA object parameters
    hipResourceDesc resDesc;
    memset(&resDesc, 0, sizeof(resDesc));
    resDesc.resType = hipResourceTypeArray;
    resDesc.res.array.array = cuArrayFrameBGRA;

    hipTextureDesc texDesc;
    memset(&texDesc, 0, sizeof(texDesc));
    texDesc.addressMode[0] = hipAddressModeClamp;
    texDesc.addressMode[1] = hipAddressModeClamp;
    texDesc.filterMode = hipFilterModePoint;
    texDesc.readMode = hipReadModeElementType;
    texDesc.normalizedCoords = false;
    cudaErrorCheck(hipCreateTextureObject(&oTexFrameBGRA, &resDesc, &texDesc, NULL));

    // surface frameLab
    hipResourceDesc rescDescFrameLab;
    memset(&rescDescFrameLab, 0, sizeof(rescDescFrameLab));
    rescDescFrameLab.resType = hipResourceTypeArray;

    rescDescFrameLab.res.array.array = cuArrayFrameLab;
    cudaErrorCheck(hipCreateSurfaceObject(&oSurfFrameLab, &rescDescFrameLab));

    // surface labels
    hipResourceDesc resDescLabels;
    memset(&resDescLabels, 0, sizeof(resDescLabels));
    resDescLabels.resType = hipResourceTypeArray;

    resDescLabels.res.array.array = cuArrayLabels;
    cudaErrorCheck(hipCreateSurfaceObject(&oSurfLabels, &resDescLabels));

    // buffers clusters , accAtt
    cudaErrorCheck(hipMalloc((void**)&d_fClusters, m_nbSpx*sizeof(float) * 5)); // 5-D centroid
    cudaErrorCheck(hipMalloc((void**)&d_fAccAtt, m_nbSpx*sizeof(float) * 6)); // 5-D centroid acc + 1 counter
    hipMemset(d_fAccAtt, 0, m_nbSpx*sizeof(float) * 6);//initialize accAtt to 0

}


void SLIC::uploadFrame(const Mat& frameBGR) {
    cv::Mat frameBGRA;
    cv::cvtColor(frameBGR, frameBGRA, CV_BGR2BGRA);
    CV_Assert(frameBGRA.type() == CV_8UC4);
    CV_Assert(frameBGRA.isContinuous());
    cudaErrorCheck(hipMemcpyToArray(cuArrayFrameBGRA, 0, 0, (uchar*)frameBGRA.data, m_nbPx* sizeof(uchar4), hipMemcpyHostToDevice));


    /*uchar* dst = new uchar[4 * m_nbPx];
    hipMemcpyFromArray(dst, cuArrayFrameBGRA, 0, 0, m_nbPx*sizeof(uchar4), hipMemcpyDeviceToHost);
    Mat matDst(m_FrameHeight, m_FrameWidth, CV_8UC4, dst);
    cout << matDst << endl;*/
}

void SLIC::gpuRGBA2Lab() {
    const int blockW = 16;
    const int blockH = blockW;
    CV_Assert(blockW*blockH <= m_deviceProp.maxThreadsPerBlock);
    dim3 threadsPerBlock(blockW, blockH);
    dim3 numBlocks(iDivUp(m_FrameWidth, blockW), iDivUp(m_FrameHeight, blockH));

    kRgb2CIELab << <numBlocks, threadsPerBlock >> >(oTexFrameBGRA, oSurfFrameLab, m_FrameWidth, m_FrameHeight);

    /*float* dst = new float[4 * m_nbPx];
    hipMemcpyFromArray(dst, cuArrayFrameLab, 0, 0, m_nbPx*sizeof(float4), hipMemcpyDeviceToHost);
    Mat matDst(m_FrameHeight, m_FrameWidth, CV_32FC4, dst);
    cout << matDst << endl;*/
}



void SLIC::gpuInitClusters() {
    int blockW = 16;
    dim3 threadsPerBlock(blockW);
    dim3 numBlocks(iDivUp(m_nbSpx, blockW));

    kInitClusters << <numBlocks, threadsPerBlock >> >(oSurfFrameLab,
        d_fClusters,
        m_FrameWidth,
        m_FrameHeight,
        m_FrameWidth / m_SpxWidth,
        m_FrameHeight / m_SpxHeight);

    /*float* fTmp = new float[m_nbSpx * 5];
    hipMemcpy(fTmp, d_fClusters, m_nbSpx * 5 * sizeof(float), hipMemcpyDeviceToHost);
    Mat matTmp(1, m_nbSpx*5, CV_32F, fTmp);
    cout << matTmp << endl;*/
}

void SLIC::assignment(){
    int hMax = m_deviceProp.maxThreadsPerBlock / m_SpxHeight;
    int nBlockPerClust = iDivUp(m_SpxHeight, hMax);

    dim3 blockPerGrid(m_nbSpx, nBlockPerClust);
    dim3 threadPerBlock(m_SpxWidth, std::min(m_SpxHeight, hMax));

    CV_Assert(threadPerBlock.x >= 3 && threadPerBlock.y >= 3);

    float wc2 = m_wc * m_wc;

    kAssignment << < blockPerGrid, threadPerBlock >> >(oSurfFrameLab,
        d_fClusters,
        m_FrameWidth,
        m_FrameHeight,
        m_SpxWidth,
        m_SpxHeight,
        wc2,
        oSurfLabels,
        d_fAccAtt);
}

void SLIC::update(){
    dim3 threadsPerBlock(m_deviceProp.maxThreadsPerBlock);
    dim3 numBlocks(iDivUp(m_nbSpx, m_deviceProp.maxThreadsPerBlock));
    kUpdate << <numBlocks, threadsPerBlock >> >(m_nbSpx, d_fClusters, d_fAccAtt);
}

void SLIC::downloadLabels(){
    hipMemcpyFromArray(h_fLabels, cuArrayLabels, 0, 0, m_nbPx* sizeof(float), hipMemcpyDeviceToHost);
}

int SLIC::enforceConnectivity() {
    int label = 0, adjlabel = 0;
    int lims = (m_FrameWidth * m_FrameHeight) / (m_nbSpx);
    lims = lims >> 2;

    const int dx4[4] = { -1, 0, 1, 0 };
    const int dy4[4] = { 0, -1, 0, 1 };

    vector<vector<int> >newLabels;
    for (int i = 0; i < m_FrameHeight; i++) {
        vector<int> nv(m_FrameWidth, -1);
        newLabels.push_back(nv);
    }

    for (int i = 0; i < m_FrameHeight; i++) {
        for (int j = 0; j < m_FrameWidth; j++){
            if (newLabels[i][j] == -1){
                vector<cv::Point> elements;
                elements.push_back(cv::Point(j, i));
                for (int k = 0; k < 4; k++){
                    int x = elements[0].x + dx4[k], y = elements[0].y + dy4[k];
                    if (x >= 0 && x < m_FrameWidth && y >= 0 && y < m_FrameHeight){
                        if (newLabels[y][x] >= 0){
                            adjlabel = newLabels[y][x];
                        }
                    }
                }
                int count = 1;
                for (int c = 0; c < count; c++){
                    for (int k = 0; k < 4; k++){
                        int x = elements[c].x + dx4[k], y = elements[c].y + dy4[k];
                        if (x >= 0 && x < m_FrameWidth && y >= 0 && y < m_FrameHeight){
                            if (newLabels[y][x] == -1 && h_fLabels[i*m_FrameWidth + j] == h_fLabels[y*m_FrameWidth + x]){
                                elements.push_back(cv::Point(x, y));
                                newLabels[y][x] = label;//m_labels[i][j];
                                count += 1;
                            }
                        }
                    }
                }
                if (count <= lims) {
                    for (int c = 0; c < count; c++) {
                        newLabels[elements[c].y][elements[c].x] = adjlabel;
                    }
                    label -= 1;
                }
                label += 1;
            }
        }
    }
    int nbSpxNoOrphan = label; // new number of spx
    for (int i = 0; i < newLabels.size(); i++)
        for (int j = 0; j < newLabels[i].size(); j++)
            h_fLabels[i*m_FrameWidth + j] = (float)newLabels[i][j];

    return nbSpxNoOrphan;
}


void SLIC::displayBound(cv::Mat& image, const float* labels, const cv::Scalar colour){
    const int dx8[8] = { -1, -1, 0, 1, 1, 1, 0, -1 };
    const int dy8[8] = { 0, -1, -1, -1, 0, 1, 1, 1 };

    /* Initialize the contour vector and the matrix detailing whether a pixel
    * is already taken to be a contour. */
    vector<cv::Point> contours;
    vector<vector<bool> > istaken;
    for (int i = 0; i < image.rows; i++) {
        vector<bool> nb;
        for (int j = 0; j < image.cols; j++) {
            nb.push_back(false);
        }
        istaken.push_back(nb);
    }
    // Go through all the pixels.
    for (int i = 0; i<image.rows; i++) {
        for (int j = 0; j < image.cols; j++) {

            int nr_p = 0;
            // Compare the pixel to its 8 neighbours.
            for (int k = 0; k < 8; k++) {
                int x = j + dx8[k], y = i + dy8[k];

                if (x >= 0 && x < image.cols && y >= 0 && y < image.rows) {
                    if (istaken[y][x] == false && labels[i*image.cols + j] != labels[y*image.cols + x]) {
                        nr_p += 1;
                    }
                }
            }
            /* Add the pixel to the contour list if desired. */
            if (nr_p >= 2) {
                contours.push_back(cv::Point(j, i));
                istaken[i][j] = true;
            }

        }
    }
    // Draw the contour pixels.
    for (int i = 0; i < (int)contours.size(); i++) {
        image.at<cv::Vec3b>(contours[i].y, contours[i].x) = cv::Vec3b((uchar)colour[0], (uchar)colour[1], (uchar)colour[2]);
    }
}
static void getSpxSizeFromDiam(const int imWidth, const int imHeight, const int diamSpx, int* spxWidth, int* spxHeight){
    int wl1, wl2;
    int hl1, hl2;
    wl1 = wl2 = diamSpx;
    hl1 = hl2 = diamSpx;

    while ((imWidth%wl1) != 0) {
        wl1++;
    }
    while ((imWidth%wl2) != 0) {
        wl2--;
    }
    while ((imHeight%hl1) != 0) {
        hl1++;
    }

    while ((imHeight%hl2) != 0) {
        hl2--;
    }
    *spxWidth = ((diamSpx - wl2) < (wl1 - diamSpx)) ? wl2 : wl1;
    *spxHeight = ((diamSpx - hl2) < (hl1 - diamSpx)) ? hl2 : hl1;
}